#include "hip/hip_runtime.h"
/**
 * @author      Christoph Schaefer cm.schaefer@gmail.com
 *
 * @section     LICENSE
 * Copyright (c) 2019 Christoph Schaefer
 *
 * This file is part of miluphcuda.
 *
 * miluphcuda is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * miluphcuda is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with miluphcuda.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "miluph.h"
#include "memory_handling.h"
#include "aneos.h"


/* allocate memory on the device for pointmasses */
int allocate_pointmass_memory(struct Pointmass *a, int allocate_immutables)
{
    int rc = 0;

	cudaVerify(hipMalloc((void**)&a->x, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vx, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_ax, memorySizeForPointmasses));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->y, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vy, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_ay, memorySizeForPointmasses));
# if DIM > 2
	cudaVerify(hipMalloc((void**)&a->z, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->vz, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feedback_az, memorySizeForPointmasses));
# endif
#endif
	cudaVerify(hipMalloc((void**)&a->m, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->rmin, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->rmax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&a->feels_particles, integermemorySizeForPointmasses));

    return rc;
}



/* allocate memory on the device for particles */
int allocate_particles_memory(struct Particle *a, int allocate_immutables)
{
    int rc = 0;

#if TENSORIAL_CORRECTION
    // also moved to p_device only
//	cudaVerify(hipMalloc((void**)&a->tensorialCorrectionMatrix, memorySizeForStress));
    // not needed anymore, let's save memory --- tschakka!
/*    if (allocate_immutables) {
        cudaVerify(hipMalloc((void**)&a->tensorialCorrectiondWdrr, MAX_NUM_INTERACTIONS * maxNumberOfParticles * sizeof(double)));
    } */
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipMalloc((void**)&a->dedt, memorySizeForParticles));
#endif

#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipMalloc((void**)&a->muijmax, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&a->drhodt, memorySizeForParticles));

#if SOLID
	cudaVerify(hipMalloc((void**)&a->S, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&a->dSdt, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&a->local_strain, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->ep, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->edotp, memorySizeForParticles));
#endif

#if NAVIER_STOKES
	cudaVerify(hipMalloc((void**)&a->Tshear, memorySizeForStress));
#endif

#if INVISCID_SPH
	cudaVerify(hipMalloc((void**)&a->beta, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->beta_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->divv_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dbetadt, memorySizeForParticles));
#endif

#if FRAGMENTATION
	memorySizeForActivationThreshold = maxNumberOfParticles * MAX_NUM_FLAWS * sizeof(double);
	cudaVerify(hipMalloc((void**)&a->d, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->damage_total, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dddt, memorySizeForParticles));

	cudaVerify(hipMalloc((void**)&a->numFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&a->numActiveFlaws, memorySizeForInteractions));
    if (allocate_immutables) {
	    cudaVerify(hipMalloc((void**)&a->flaws, memorySizeForActivationThreshold));
    }
# if PALPHA_POROSITY
	cudaVerify(hipMalloc((void**)&a->damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->ddamage_porjutzidt, memorySizeForParticles));
# endif
#endif

    if (allocate_immutables) {
        cudaVerify(hipMalloc((void**)&a->h0, memorySizeForParticles));
    }

#if GHOST_BOUNDARIES
	cudaVerify(hipMalloc((void**)&a->real_partner, memorySizeForInteractions));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipMalloc((void**)&a->pold, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dalphadrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->delpdelrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->delpdele, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->f, memorySizeForParticles));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipMalloc((void**)&a->compressive_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->tensile_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->shear_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->K, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_0prime, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_c_plus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_c_minus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&a->flag_plastic, memorySizeForInteractions));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMalloc((void**)&a->alpha_epspor, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->dalpha_epspordt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->depsilon_vdt, memorySizeForParticles));
#endif

    cudaVerify(hipMalloc((void**)&a->x0, memorySizeForTree));
#if DIM > 1
    cudaVerify(hipMalloc((void**)&a->y0, memorySizeForTree));
#if DIM > 2
    cudaVerify(hipMalloc((void**)&a->z0, memorySizeForTree));
#endif
#endif
	cudaVerify(hipMalloc((void**)&a->x, memorySizeForTree));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->y, memorySizeForTree));
#endif
	cudaVerify(hipMalloc((void**)&a->vx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->vy, memorySizeForParticles));
#endif
	cudaVerify(hipMalloc((void**)&a->dxdt, memorySizeForParticles));
#if DIM > 1
 	cudaVerify(hipMalloc((void**)&a->dydt, memorySizeForParticles));
#endif

#if XSPH
	cudaVerify(hipMalloc((void**)&a->xsphvx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->xsphvy, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMalloc((void**)&a->ax, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_ax, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&a->ay, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_ay, memorySizeForParticles));
#endif
	cudaVerify(hipMalloc((void**)&a->m, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&a->h, memorySizeForParticles));
#if INTEGRATE_SML
	cudaVerify(hipMalloc((void**)&a->dhdt, memorySizeForParticles));
#endif

#if SML_CORRECTION
	cudaVerify(hipMalloc((void**)&a->sml_omega, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&a->rho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->p, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->e, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->cs, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->noi, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&a->depth, memorySizeForInteractions));
#if MORE_OUTPUT
	cudaVerify(hipMalloc((void**)&a->p_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->p_max, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->rho_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->e_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->e_max, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->cs_min, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&a->cs_max, memorySizeForParticles));
#endif
// moved to p_device only, so we don't need mem here anymore
//	cudaVerify(hipMalloc((void**)&a->materialId, memorySizeForInteractions));

#if JC_PLASTICITY
	cudaVerify(hipMalloc((void**)&a->T, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->dTdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->jc_f, memorySizeForParticles));
#endif

#if DIM > 2
	cudaVerify(hipMalloc((void**)&a->z, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&a->dzdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->vz, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->az, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&a->g_az, memorySizeForParticles));
#if XSPH
	cudaVerify(hipMalloc((void**)&a->xsphvz, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMemset(a->ax, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_ax, 0, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMemset(a->ay, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_ay, 0, memorySizeForParticles));
#if DIM == 3
	cudaVerify(hipMemset(a->az, 0, memorySizeForParticles));
	cudaVerify(hipMemset(a->g_az, 0, memorySizeForParticles));
#endif
#endif

    return rc;
}



int copy_gravitational_accels_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->g_ax, src->g_ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->g_ay, src->g_ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 2
    cudaVerify(hipMemcpy(dst->g_az, src->g_az, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

    return rc;
}



int copy_pointmass_derivatives_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->ax, src->ax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_ax, src->feedback_ax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->ay, src->ay, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_ay, src->feedback_ay, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# if DIM > 2
    cudaVerify(hipMemcpy(dst->az, src->az, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->feedback_az, src->feedback_az, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# endif
#endif

    return rc;
}



int copy_particles_derivatives_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy(dst->ax, src->ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_ax, src->g_ax, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dxdt, src->dxdt, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if DIM > 1
    cudaVerify(hipMemcpy(dst->ay, src->ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_ay, src->g_ay, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dydt, src->dydt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if DIM > 2
    cudaVerify(hipMemcpy(dst->az, src->az, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->g_az, src->g_az, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dzdt, src->dzdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

    cudaVerify(hipMemcpy(dst->drhodt, src->drhodt, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if INTEGRATE_SML
    cudaVerify(hipMemcpy(dst->dhdt, src->dhdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SML_CORRECTION
    cudaVerify(hipMemcpy(dst->sml_omega, src->sml_omega, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if PALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->dalphadt, src->dalphadt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->ddamage_porjutzidt, src->ddamage_porjutzidt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->dalpha_epspordt, src->dalpha_epspordt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->depsilon_vdt, src->depsilon_vdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if INTEGRATE_ENERGY
    cudaVerify(hipMemcpy(dst->dedt, src->dedt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SOLID
    cudaVerify(hipMemcpy(dst->dSdt, src->dSdt, memorySizeForStress, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->edotp, src->edotp, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if INVISCID_SPH
	cudaVerify(hipMemcpy(dst->dbetadt, src->dbetadt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if JC_PLASTICITY
    cudaVerify(hipMemcpy(dst->dTdt, src->dTdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->dddt, src->dddt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->numActiveFlaws, src->numActiveFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



int copy_pointmass_immutables_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy((*dst).m, (*src).m, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).feels_particles, (*src).feels_particles, integermemorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).rmin, (*src).rmin, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).rmax, (*src).rmax, memorySizeForPointmasses, hipMemcpyDeviceToDevice));

    return rc;
}



int copy_particles_immutables_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy((*dst).x0, (*src).x0, memorySizeForTree, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy((*dst).y0, (*src).y0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif
#if DIM > 2
    cudaVerify(hipMemcpy((*dst).z0, (*src).z0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif
    cudaVerify(hipMemcpy((*dst).m, (*src).m, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).h, (*src).h, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy((*dst).cs, (*src).cs, memorySizeForParticles, hipMemcpyDeviceToDevice));
    //cudaVerify(hipMemcpy((*dst).materialId, (*src).materialId, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#if FRAGMENTATION
	cudaVerify(hipMemcpy(dst->numFlaws, src->numFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
    //cudaVerify(hipMemcpy(dst->flaws, src->flaws, memorySizeForActivationThreshold, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



int copy_pointmass_variables_device_to_device(struct Pointmass *dst, struct Pointmass *src)
{
    int rc = 0;
    cudaVerify(hipMemcpy(dst->x, src->x, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    // mass is variable
    cudaVerify(hipMemcpy(dst->m, src->m, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->y, src->y, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# if DIM > 2
    cudaVerify(hipMemcpy(dst->z, src->z, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForPointmasses, hipMemcpyDeviceToDevice));
# endif
#endif

    return rc;
}



int copy_particles_variables_device_to_device(struct Particle *dst, struct Particle *src)
{
    int rc = 0;

    cudaVerify(hipMemcpy(dst->x, src->x, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->x0, src->x0, memorySizeForTree, hipMemcpyDeviceToDevice));
    // materialId moved to p_device aka p_rhs only
    //cudaVerify(hipMemcpy((*dst).materialId, (*src).materialId, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#if DIM > 1
    cudaVerify(hipMemcpy(dst->y, src->y, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->y0, src->y0, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vy, src->vy, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#if DIM > 2
    cudaVerify(hipMemcpy(dst->z0, src->z0, memorySizeForTree, hipMemcpyDeviceToDevice));
#endif

    cudaVerify(hipMemcpy(dst->vx, src->vx, memorySizeForParticles, hipMemcpyDeviceToDevice));

    cudaVerify(hipMemcpy(dst->rho, src->rho, memorySizeForParticles, hipMemcpyDeviceToDevice));

    cudaVerify(hipMemcpy(dst->h, src->h, memorySizeForParticles, hipMemcpyDeviceToDevice));

#if INTEGRATE_ENERGY
    cudaVerify(hipMemcpy(dst->e, src->e, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if PALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->alpha_jutzi, src->alpha_jutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->alpha_jutzi_old, src->alpha_jutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalphadp, src->dalphadp, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalphadrho, src->dalphadrho, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dp, src->dp, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->delpdelrho, src->delpdelrho, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->delpdele, src->delpdele, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->f, src->f, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->p, src->p, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->pold, src->pold, memorySizeForParticles, hipMemcpyDeviceToDevice));
# if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->damage_porjutzi, src->damage_porjutzi, memorySizeForParticles, hipMemcpyDeviceToDevice));
# endif
#endif

#if MORE_OUTPUT
    cudaVerify(hipMemcpy(dst->p_min, src->p_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->p_max, src->p_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_min, src->rho_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_max, src->rho_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->e_min, src->e_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->e_max, src->e_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->cs_min, src->cs_min, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->cs_max, src->cs_max, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipMemcpy(dst->compressive_strength, src->compressive_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->tensile_strength, src->tensile_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->shear_strength, src->shear_strength, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->K, src->K, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_0prime, src->rho_0prime, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_c_plus, src->rho_c_plus, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->rho_c_minus, src->rho_c_minus, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->flag_rho_0prime, src->flag_rho_0prime, memorySizeForInteractions, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->flag_plastic, src->flag_plastic, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(dst->alpha_epspor, src->alpha_epspor, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->dalpha_epspordt, src->dalpha_epspordt, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->epsilon_v, src->epsilon_v, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->depsilon_vdt, src->depsilon_vdt, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if DIM > 2
    cudaVerify(hipMemcpy(dst->z, src->z, memorySizeForTree, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->vz, src->vz, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#if SOLID
    cudaVerify(hipMemcpy(dst->S, src->S, memorySizeForStress, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->ep, src->ep, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif
#if NAVIER_STOKES
    cudaVerify(hipMemcpy(dst->Tshear, src->Tshear, memorySizeForStress, hipMemcpyDeviceToDevice));
#endif

#if INVISCID_SPH
    cudaVerify(hipMemcpy(dst->beta, src->beta, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->beta_old, src->beta_old, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->divv_old, src->divv_old, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if JC_PLASTICITY
    cudaVerify(hipMemcpy(dst->T, src->T, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->jc_f, src->jc_f, memorySizeForParticles, hipMemcpyDeviceToDevice));
#endif

#if FRAGMENTATION
    cudaVerify(hipMemcpy(dst->d, src->d, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->damage_total, src->damage_total, memorySizeForParticles, hipMemcpyDeviceToDevice));
    cudaVerify(hipMemcpy(dst->numActiveFlaws, src->numActiveFlaws, memorySizeForInteractions, hipMemcpyDeviceToDevice));
#endif

    return rc;
}



/* free runge-kutta memory for pointmasses on the device */
int free_pointmass_memory(struct Pointmass *a, int free_immutables)
{
    int rc = 0;
	cudaVerify(hipFree(a->x));
	cudaVerify(hipFree(a->vx));
	cudaVerify(hipFree(a->ax));
	cudaVerify(hipFree(a->feedback_ax));
	cudaVerify(hipFree(a->m));
	cudaVerify(hipFree(a->feels_particles));
	cudaVerify(hipFree(a->rmin));
	cudaVerify(hipFree(a->rmax));
#if DIM > 1
	cudaVerify(hipFree(a->y));
	cudaVerify(hipFree(a->vy));
	cudaVerify(hipFree(a->ay));
	cudaVerify(hipFree(a->feedback_ay));
# if DIM > 2
	cudaVerify(hipFree(a->z));
	cudaVerify(hipFree(a->vz));
	cudaVerify(hipFree(a->az));
	cudaVerify(hipFree(a->feedback_az));
# endif
#endif

    return rc;
}



/* free runge-kutta memory on the device */
int free_particles_memory(struct Particle *a, int free_immutables)
{
    int rc = 0;

	cudaVerify(hipFree(a->x));
	cudaVerify(hipFree(a->x0));
	cudaVerify(hipFree(a->dxdt));
	cudaVerify(hipFree(a->vx));
	cudaVerify(hipFree(a->ax));
	cudaVerify(hipFree(a->g_ax));
	cudaVerify(hipFree(a->m));
#if DIM > 1
	cudaVerify(hipFree(a->dydt));
	cudaVerify(hipFree(a->y));
	cudaVerify(hipFree(a->y0));
	cudaVerify(hipFree(a->vy0));
	cudaVerify(hipFree(a->vy));
	cudaVerify(hipFree(a->ay));
	cudaVerify(hipFree(a->g_ay));
#endif

#if XSPH
	cudaVerify(hipFree(a->xsphvx));
#if DIM > 1
	cudaVerify(hipFree(a->xsphvy));
#endif
#endif
	cudaVerify(hipFree(a->h));
	cudaVerify(hipFree(a->rho));
	cudaVerify(hipFree(a->p));
	cudaVerify(hipFree(a->e));
	cudaVerify(hipFree(a->cs));
	cudaVerify(hipFree(a->noi));
	cudaVerify(hipFree(a->depth));
#if MORE_OUTPUT
	cudaVerify(hipFree(a->p_min));
	cudaVerify(hipFree(a->p_max));
	cudaVerify(hipFree(a->rho_min));
	cudaVerify(hipFree(a->rho_max));
	cudaVerify(hipFree(a->e_min));
	cudaVerify(hipFree(a->e_max));
	cudaVerify(hipFree(a->cs_min));
	cudaVerify(hipFree(a->cs_max));
#endif
    // materialId only on p_device
	//cudaVerify(hipFree(a->materialId));
#if DIM > 2
	cudaVerify(hipFree(a->z));
	cudaVerify(hipFree(a->z0));
	cudaVerify(hipFree(a->dzdt));
	cudaVerify(hipFree(a->vz));
#if XSPH
	cudaVerify(hipFree(a->xsphvz));
#endif
	cudaVerify(hipFree(a->az));
	cudaVerify(hipFree(a->g_az));
#endif


#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipFree(a->muijmax));
#endif
#if (NAVIER_STOKES || BALSARA_SWITCH || INVISCID_SPH || INTEGRATE_ENERGY)
	cudaVerify(hipFree(a->divv));
	cudaVerify(hipFree(a->curlv));
#endif

#if INVISCID_SPH
	cudaVerify(hipFree(a->beta));
	cudaVerify(hipFree(a->beta_old));
	cudaVerify(hipFree(a->divv_old));
	cudaVerify(hipFree(a->dbetadt));
#endif

#if TENSORIAL_CORRECTION
	//cudaVerify(hipFree(a->tensorialCorrectionMatrix));
    /*
    if (free_immutables) {
	    cudaVerify(hipFree(a->tensorialCorrectiondWdrr));
    } */
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipFree(a->dedt));
#endif
#if GHOST_BOUNDARIES
	cudaVerify(hipFree(a->real_partner));
#endif

	cudaVerify(hipFree(a->drhodt));

#if INTEGRATE_SML
	cudaVerify(hipFree(a->dhdt));
#endif

#if SML_CORRECTION
    cudaVerify(hipFree(a->sml_omega));
#endif

#if SOLID
	cudaVerify(hipFree(a->S));
	cudaVerify(hipFree(a->dSdt));
	cudaVerify(hipFree(a->local_strain));
    cudaVerify(hipFree(a->ep));
    cudaVerify(hipFree(a->edotp));
#endif
#if NAVIER_STOKES
	cudaVerify(hipFree(a->Tshear));
#endif

#if JC_PLASTICITY
	cudaVerify(hipFree(a->T));
	cudaVerify(hipFree(a->dTdt));
	cudaVerify(hipFree(a->jc_f));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipFree(a->pold));
	cudaVerify(hipFree(a->alpha_jutzi));
	cudaVerify(hipFree(a->alpha_jutzi_old));
	cudaVerify(hipFree(a->dalphadt));
	cudaVerify(hipFree(a->f));
	cudaVerify(hipFree(a->dalphadp));
	cudaVerify(hipFree(a->dp));
	cudaVerify(hipFree(a->delpdelrho));
	cudaVerify(hipFree(a->delpdele));
	cudaVerify(hipFree(a->dalphadrho));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipFree(a->compressive_strength));
    cudaVerify(hipFree(a->tensile_strength));
    cudaVerify(hipFree(a->shear_strength));
    cudaVerify(hipFree(a->K));
    cudaVerify(hipFree(a->rho_0prime));
    cudaVerify(hipFree(a->rho_c_plus));
    cudaVerify(hipFree(a->rho_c_minus));
    cudaVerify(hipFree(a->flag_rho_0prime));
    cudaVerify(hipFree(a->flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipFree(a->alpha_epspor));
    cudaVerify(hipFree(a->dalpha_epspordt));
    cudaVerify(hipFree(a->epsilon_v));
    cudaVerify(hipFree(a->depsilon_vdt));
#endif

#if FRAGMENTATION
	cudaVerify(hipFree(a->d));
	cudaVerify(hipFree(a->damage_total));
	cudaVerify(hipFree(a->dddt));
	cudaVerify(hipFree(a->numFlaws));
	cudaVerify(hipFree(a->numActiveFlaws));
    if (free_immutables) {
	    cudaVerify(hipFree(a->flaws));
    }
    if (free_immutables) {
	    cudaVerify(hipFree(a->h0));
    }
# if PALPHA_POROSITY
	cudaVerify(hipFree(a->damage_porjutzi));
	cudaVerify(hipFree(a->ddamage_porjutzidt));
# endif
#endif

    return rc;
}



/* allocate memory for tree and basic particle struct */
int init_allocate_memory(void)
{
    int rc = 0;

	numberOfNodes = ceil(2.5 * maxNumberOfParticles);
    if (numberOfNodes < 1024*numberOfMultiprocessors)
        numberOfNodes = 1024*numberOfMultiprocessors;

#define WARPSIZE 32
    
    while ((numberOfNodes & (WARPSIZE-1)) != 0)
        numberOfNodes++;

	if (param.verbose) {
        fprintf(stdout, "\nAllocating memory for %d particles...\n", numberOfParticles);
	    fprintf(stdout, "Allocating memory for %d pointmasses...\n", numberOfPointmasses);
        fprintf(stdout, "Number of nodes of tree: %d\n", numberOfNodes);
    }

	memorySizeForParticles = maxNumberOfParticles * sizeof(double);
	memorySizeForPointmasses = numberOfPointmasses * sizeof(double);
	integermemorySizeForPointmasses = numberOfPointmasses * sizeof(int);
	memorySizeForTree = numberOfNodes * sizeof(double);
	memorySizeForStress = maxNumberOfParticles * DIM * DIM * sizeof(double);
	memorySizeForChildren = numberOfChildren * (numberOfNodes-numberOfRealParticles) * sizeof(int);
	memorySizeForInteractions = maxNumberOfParticles * sizeof(int);

    cudaVerify(hipHostMalloc((void**)&p_host.x, memorySizeForTree));
	cudaVerify(hipHostMalloc((void**)&p_host.vx, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ax, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.g_ax, memorySizeForParticles));
#if DIM > 1
    cudaVerify(hipHostMalloc((void**)&p_host.y, memorySizeForTree));
	cudaVerify(hipHostMalloc((void**)&p_host.vy, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ay, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.g_ay, memorySizeForParticles));
#endif
#if DIM > 2
    cudaVerify(hipHostMalloc((void**)&p_host.z, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vz, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.az, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.g_az, memorySizeForParticles));
#endif
    cudaVerify(hipHostMalloc((void**)&p_host.m, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.h, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.p, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.e, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.cs, memorySizeForParticles));

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipHostMalloc((void**)&pointmass_host.x, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vx, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.x, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vx, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.ax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_ax, memorySizeForPointmasses));
#if DIM > 1
	cudaVerify(hipHostMalloc((void**)&pointmass_host.y, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vy, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.y, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vy, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.ay, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_ay, memorySizeForPointmasses));
#if DIM > 2
	cudaVerify(hipHostMalloc((void**)&pointmass_host.z, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.vz, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.z, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.vz, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.az, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feedback_az, memorySizeForPointmasses));
#endif
#endif
	cudaVerify(hipHostMalloc((void**)&pointmass_host.rmin, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.rmax, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.rmin, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.rmax, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.m, memorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.m, memorySizeForPointmasses));
	cudaVerify(hipHostMalloc((void**)&pointmass_host.feels_particles, integermemorySizeForPointmasses));
	cudaVerify(hipMalloc((void**)&pointmass_device.feels_particles, integermemorySizeForPointmasses));
#endif

#if MORE_OUTPUT
	cudaVerify(hipHostMalloc((void**)&p_host.p_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.p_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.rho_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.rho_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.e_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.e_max, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.cs_min, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.cs_max, memorySizeForParticles));
#endif

	cudaVerify(hipHostMalloc((void**)&p_host.noi, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.depth, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&interactions_host, memorySizeForInteractions*MAX_NUM_INTERACTIONS));
	cudaVerify(hipHostMalloc((void**)&p_host.materialId, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&childList_host, memorySizeForChildren));

#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipMalloc((void**)&p_device.muijmax, memorySizeForParticles));
#endif

#if (NAVIER_STOKES || BALSARA_SWITCH || INVISCID_SPH || INTEGRATE_ENERGY)
	cudaVerify(hipMalloc((void**)&p_device.divv, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.curlv, memorySizeForParticles*DIM));
#endif

#if INVISCID_SPH
	cudaVerify(hipMalloc((void**)&p_device.beta, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.beta_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.divv_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dbetadt, memorySizeForParticles));
#endif

#if TENSORIAL_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.tensorialCorrectionMatrix, memorySizeForStress));
	//cudaVerify(hipMalloc((void**)&p_device.tensorialCorrectiondWdrr, MAX_NUM_INTERACTIONS * maxNumberOfParticles * sizeof(double)));
#endif

#if SHEPARD_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.shepard_correction, memorySizeForParticles));
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipHostMalloc((void**)&p_host.dedt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dedt, memorySizeForParticles));
#endif

	cudaVerify(hipHostMalloc((void**)&p_host.drhodt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.drhodt, memorySizeForParticles));

#if SOLID
	cudaVerify(hipHostMalloc((void**)&p_host.S, memorySizeForStress));
	cudaVerify(hipHostMalloc((void**)&p_host.dSdt, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.S, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.dSdt, memorySizeForStress));
	cudaVerify(hipHostMalloc((void**)&p_host.local_strain, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.local_strain, memorySizeForParticles));
	cudaVerify(hipMalloc((void**) &p_device.sigma, memorySizeForStress));
    cudaVerify(hipMalloc((void**)&p_device.plastic_f, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.ep, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.ep, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.edotp, memorySizeForParticles));
#endif

#if NAVIER_STOKES
	cudaVerify(hipHostMalloc((void**)&p_host.Tshear, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.Tshear, memorySizeForStress));
	cudaVerify(hipMalloc((void**)&p_device.eta, memorySizeForParticles));
#endif

#if ARTIFICIAL_STRESS
	cudaVerify(hipMalloc((void**) &p_device.R, memorySizeForStress));
#endif

#if JC_PLASTICITY
	cudaVerify(hipMalloc((void**)&p_device.T, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.T, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dTdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.jc_f, memorySizeForParticles));
#endif

#if FRAGMENTATION
	memorySizeForActivationThreshold = maxNumberOfParticles * MAX_NUM_FLAWS * sizeof(double);
	cudaVerify(hipHostMalloc((void**)&p_host.d, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.dddt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.d, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.damage_total, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dddt, memorySizeForParticles));

	cudaVerify(hipHostMalloc((void**)&p_host.numFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.numFlaws, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.numActiveFlaws, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.numActiveFlaws, memorySizeForInteractions));
	cudaVerify(hipHostMalloc((void**)&p_host.flaws, memorySizeForActivationThreshold));
	cudaVerify(hipMalloc((void**)&p_device.flaws, memorySizeForActivationThreshold));
# if PALPHA_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.ddamage_porjutzidt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.damage_porjutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.ddamage_porjutzidt, memorySizeForParticles));
# endif
#endif

	cudaVerify(hipMalloc((void**)&p_device.h0, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.h0, memorySizeForParticles));

#if GHOST_BOUNDARIES
	cudaVerify(hipMalloc((void**)&p_device.real_partner, memorySizeForInteractions));
#endif

#if PALPHA_POROSITY
	cudaVerify(hipHostMalloc((void**)&p_host.alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipHostMalloc((void**)&p_host.pold, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.pold, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.alpha_jutzi, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.alpha_jutzi_old, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dp, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dalphadrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.f, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.delpdelrho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.delpdele, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_old, memorySizeForParticles));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.compressive_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.tensile_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.shear_strength, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_0prime, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_c_plus, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.rho_c_minus, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.K, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipHostMalloc((void**)&p_host.flag_plastic, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&p_device.compressive_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.tensile_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.shear_strength, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.K, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_0prime, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_c_plus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.rho_c_minus, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.flag_rho_0prime, memorySizeForInteractions));
    cudaVerify(hipMalloc((void**)&p_device.flag_plastic, memorySizeForInteractions));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipHostMalloc((void**)&p_host.alpha_epspor, memorySizeForParticles));
    cudaVerify(hipHostMalloc((void**)&p_host.epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.alpha_epspor, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.dalpha_epspordt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.epsilon_v, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.depsilon_vdt, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.x, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_x, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_local_cellsize, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vx, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dxdt, memorySizeForParticles));

#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.y, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_y, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vy, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dydt, memorySizeForParticles));
    cudaVerify(hipMalloc((void**)&p_device.y0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vy0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vy0, memorySizeForTree));
#endif

    cudaVerify(hipMalloc((void**)&p_device.x0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vx0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vx0, memorySizeForTree));
#if DIM > 2
    cudaVerify(hipMalloc((void**)&p_device.z0, memorySizeForTree));
    cudaVerify(hipMalloc((void**)&p_device.vz0, memorySizeForTree));
    cudaVerify(hipHostMalloc((void**)&p_host.vz0, memorySizeForTree));
#endif

#if XSPH
	cudaVerify(hipMalloc((void**)&p_device.xsphvx, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.xsphvy, memorySizeForParticles));
#endif
#endif
	cudaVerify(hipMalloc((void**)&p_device.ax, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_ax, memorySizeForParticles));

#if DIM > 1
	cudaVerify(hipMalloc((void**)&p_device.ay, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_ay, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.m, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.h, memorySizeForParticles));

#if INTEGRATE_SML
	cudaVerify(hipMalloc((void**)&p_device.dhdt, memorySizeForParticles));
#endif

#if SML_CORRECTION
	cudaVerify(hipMalloc((void**)&p_device.sml_omega, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&p_device.rho, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.p, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.depth, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.noi, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.materialId, memorySizeForInteractions));
	cudaVerify(hipMalloc((void**)&p_device.materialId0, memorySizeForInteractions));

#if MORE_OUTPUT
	cudaVerify(hipMalloc((void**)&p_device.p_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.p_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.rho_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.rho_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.e_max, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_min, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.cs_max, memorySizeForParticles));
#endif

	cudaVerify(hipMalloc((void**)&interactions, memorySizeForInteractions*MAX_NUM_INTERACTIONS));
	cudaVerify(hipMalloc((void**)&childListd, memorySizeForChildren));
#if DIM > 2
	cudaVerify(hipMalloc((void**)&p_device.z, memorySizeForTree));
	cudaVerify(hipMalloc((void**)&p_device.g_z, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.dzdt, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.vz, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.az, memorySizeForParticles));
	cudaVerify(hipMalloc((void**)&p_device.g_az, memorySizeForParticles));
#if XSPH
	cudaVerify(hipMalloc((void**)&p_device.xsphvz, memorySizeForParticles));
#endif
#endif

	cudaVerify(hipMemset(p_device.ax, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_ax, 0, memorySizeForParticles));
#if DIM > 1
	cudaVerify(hipMemset(p_device.ay, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_ay, 0, memorySizeForParticles));
#endif
#if DIM > 2
	cudaVerify(hipMemset(p_device.az, 0, memorySizeForParticles));
	cudaVerify(hipMemset(p_device.g_az, 0, memorySizeForParticles));
#endif

    return rc;
}



int copy_particle_data_to_device()
{
    int rc = 0;

	if (param.verbose)
        fprintf(stdout, "\nCopying particle data to device...\n");

	cudaVerify(hipMemcpy(p_device.x0, p_host.x, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.x, p_host.x, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vx, p_host.vx, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vx0, p_host.vx0, memorySizeForParticles, hipMemcpyHostToDevice));
#if DIM > 1
	cudaVerify(hipMemcpy(p_device.y0, p_host.y, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.y, p_host.y, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vy, p_host.vy, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vy0, p_host.vy0, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if DIM > 2
	cudaVerify(hipMemcpy(p_device.z0, p_host.z, memorySizeForTree, hipMemcpyHostToDevice));
#endif

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipMemcpy(pointmass_device.x, pointmass_host.x, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vx, pointmass_host.vx, memorySizeForPointmasses, hipMemcpyHostToDevice));
# if DIM > 1
	cudaVerify(hipMemcpy(pointmass_device.y, pointmass_host.y, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vy, pointmass_host.vy, memorySizeForPointmasses, hipMemcpyHostToDevice));
#  if DIM > 2
	cudaVerify(hipMemcpy(pointmass_device.z, pointmass_host.z, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.vz, pointmass_host.vz, memorySizeForPointmasses, hipMemcpyHostToDevice));
#  endif
# endif
	cudaVerify(hipMemcpy(pointmass_device.rmin, pointmass_host.rmin, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.rmax, pointmass_host.rmax, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.m, pointmass_host.m, memorySizeForPointmasses, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(pointmass_device.feels_particles, pointmass_host.feels_particles, integermemorySizeForPointmasses, hipMemcpyHostToDevice));
#endif

	cudaVerify(hipMemcpy(p_device.h, p_host.h, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.cs, p_host.cs, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.m, p_host.m, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.rho, p_host.rho, memorySizeForParticles, hipMemcpyHostToDevice));
#if INTEGRATE_ENERGY
	cudaVerify(hipMemcpy(p_device.e, p_host.e, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if SOLID
	cudaVerify(hipMemcpy(p_device.S, p_host.S, memorySizeForStress, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.ep, p_host.ep, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if NAVIER_STOKES
	cudaVerify(hipMemcpy(p_device.Tshear, p_host.Tshear, memorySizeForStress, hipMemcpyHostToDevice));
#endif
#if PALPHA_POROSITY
	cudaVerify(hipMemcpy(p_device.alpha_jutzi, p_host.alpha_jutzi, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.alpha_jutzi_old, p_host.alpha_jutzi_old, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.p, p_host.p, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.pold, p_host.pold, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if MORE_OUTPUT
    cudaVerify(hipMemcpy(p_device.p_min, p_host.p_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.p_max, p_host.p_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_min, p_host.rho_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_max, p_host.rho_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.e_min, p_host.e_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.e_max, p_host.e_max, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.cs_min, p_host.cs_min, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.cs_max, p_host.cs_max, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if SIRONO_POROSITY
    cudaVerify(hipMemcpy(p_device.compressive_strength, p_host.compressive_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.tensile_strength, p_host.tensile_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.shear_strength, p_host.shear_strength, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_0prime, p_host.rho_0prime, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_c_plus, p_host.rho_c_plus, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.rho_c_minus, p_host.rho_c_minus, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.K, p_host.K, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flag_rho_0prime, p_host.flag_rho_0prime, memorySizeForInteractions, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flag_plastic, p_host.flag_plastic, memorySizeForInteractions, hipMemcpyHostToDevice));
#endif
#if EPSALPHA_POROSITY
    cudaVerify(hipMemcpy(p_device.alpha_epspor, p_host.alpha_epspor, memorySizeForParticles, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.epsilon_v, p_host.epsilon_v, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
    cudaVerify(hipMemcpy(p_device.h0, p_host.h0, memorySizeForParticles, hipMemcpyHostToDevice));
#if JC_PLASTICITY
	cudaVerify(hipMemcpy(p_device.T, p_host.T, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
#if FRAGMENTATION
	cudaVerify(hipMemcpy(p_device.d, p_host.d, memorySizeForParticles, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.numFlaws, p_host.numFlaws, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.numActiveFlaws, p_host.numActiveFlaws, memorySizeForInteractions, hipMemcpyHostToDevice));
    cudaVerify(hipMemcpy(p_device.flaws, p_host.flaws, memorySizeForActivationThreshold, hipMemcpyHostToDevice));
# if PALPHA_POROSITY
    cudaVerify(hipMemcpy(p_device.damage_porjutzi, p_host.damage_porjutzi, memorySizeForParticles, hipMemcpyHostToDevice));
# endif
#endif
	cudaVerify(hipMemcpy(p_device.noi, p_host.noi, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.materialId, p_host.materialId, memorySizeForInteractions, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.materialId0, p_host.materialId, memorySizeForInteractions, hipMemcpyHostToDevice));
#if DIM > 2
	cudaVerify(hipMemcpy(p_device.z, p_host.z, memorySizeForTree, hipMemcpyHostToDevice));
	cudaVerify(hipMemcpy(p_device.vz, p_host.vz, memorySizeForParticles, hipMemcpyHostToDevice));
#endif
	cudaVerify(hipMemset((void *) childListd, -1, memorySizeForChildren));

    return rc;
}



int free_memory()
{
    int rc = 0;

	/* free device memory */
	if (param.verbose)
        fprintf(stdout, "Freeing memory...\n");
	cudaVerify(hipFree(p_device.x));
	cudaVerify(hipFree(p_device.g_x));
	cudaVerify(hipFree(p_device.g_local_cellsize));
	cudaVerify(hipFree(p_device.depth));
	cudaVerify(hipFree(p_device.x0));
	cudaVerify(hipFree(p_device.dxdt));
	cudaVerify(hipFree(p_device.vx));
	cudaVerify(hipFree(p_device.vx0));
	cudaVerify(hipHostFree(p_host.vx0));
	cudaVerify(hipFree(p_device.ax));
	cudaVerify(hipFree(p_device.g_ax));
	cudaVerify(hipFree(p_device.m));

#if DIM > 1
	cudaVerify(hipFree(p_device.vy0));
	cudaVerify(hipHostFree(p_host.vy0));
#if DIM > 2
	cudaVerify(hipFree(p_device.vz0));
	cudaVerify(hipHostFree(p_host.vz0));
#endif
#endif
#if DIM > 1
	cudaVerify(hipFree(p_device.y));
	cudaVerify(hipFree(p_device.g_y));
	cudaVerify(hipFree(p_device.y0));
	cudaVerify(hipFree(p_device.vy));
	cudaVerify(hipFree(p_device.dydt));
	cudaVerify(hipFree(p_device.ay));
	cudaVerify(hipFree(p_device.g_ay));
#endif

#if GRAVITATING_POINT_MASSES
	cudaVerify(hipFree(pointmass_device.x));
	cudaVerify(hipFree(pointmass_device.vx));
	cudaVerify(hipFree(pointmass_device.ax));
	cudaVerify(hipFree(pointmass_device.feedback_ax));
# if DIM > 1
	cudaVerify(hipFree(pointmass_device.y));
	cudaVerify(hipFree(pointmass_device.vy));
	cudaVerify(hipFree(pointmass_device.ay));
	cudaVerify(hipFree(pointmass_device.feedback_ay));
#  if DIM > 2
	cudaVerify(hipFree(pointmass_device.z));
	cudaVerify(hipFree(pointmass_device.vz));
	cudaVerify(hipFree(pointmass_device.az));
	cudaVerify(hipFree(pointmass_device.feedback_az));
#  endif
# endif
	cudaVerify(hipFree(pointmass_device.m));
	cudaVerify(hipFree(pointmass_device.feels_particles));
	cudaVerify(hipFree(pointmass_device.rmin));
	cudaVerify(hipFree(pointmass_device.rmax));

	cudaVerify(hipHostFree(pointmass_host.x));
	cudaVerify(hipHostFree(pointmass_host.vx));
	cudaVerify(hipHostFree(pointmass_host.ax));
# if DIM > 1
	cudaVerify(hipHostFree(pointmass_host.y));
	cudaVerify(hipHostFree(pointmass_host.vy));
	cudaVerify(hipHostFree(pointmass_host.ay));
#  if DIM > 2
	cudaVerify(hipHostFree(pointmass_host.z));
	cudaVerify(hipHostFree(pointmass_host.vz));
	cudaVerify(hipHostFree(pointmass_host.az));
#  endif
# endif
	cudaVerify(hipHostFree(pointmass_host.m));
	cudaVerify(hipHostFree(pointmass_host.feels_particles));
	cudaVerify(hipHostFree(pointmass_host.rmin));
	cudaVerify(hipHostFree(pointmass_host.rmax));
#endif

#if XSPH
	cudaVerify(hipFree(p_device.xsphvx));
#if DIM > 1
	cudaVerify(hipFree(p_device.xsphvy));
#endif
#endif
	cudaVerify(hipFree(p_device.h));
	cudaVerify(hipFree(p_device.rho));
	cudaVerify(hipFree(p_device.p));
	cudaVerify(hipFree(p_device.e));
	cudaVerify(hipFree(p_device.cs));
	cudaVerify(hipFree(p_device.noi));
#if MORE_OUTPUT
	cudaVerify(hipFree(p_device.p_min));
    cudaVerify(hipFree(p_device.p_max));
    cudaVerify(hipFree(p_device.rho_min));
    cudaVerify(hipFree(p_device.rho_max));
	cudaVerify(hipFree(p_device.e_min));
    cudaVerify(hipFree(p_device.e_max));
    cudaVerify(hipFree(p_device.cs_min));
    cudaVerify(hipFree(p_device.cs_max));
#endif
#if ARTIFICIAL_VISCOSITY
	cudaVerify(hipFree(p_device.muijmax));
#endif
#if INVISCID_SPH
	cudaVerify(hipFree(p_device.beta));
	cudaVerify(hipFree(p_device.beta_old));
	cudaVerify(hipFree(p_device.divv_old));
#endif
	cudaVerify(hipFree(interactions));
	cudaVerify(hipFree(p_device.materialId));
	cudaVerify(hipFree(p_device.materialId0));
	cudaVerify(hipFree(childListd));
#if DIM > 2
	cudaVerify(hipFree(p_device.z));
	cudaVerify(hipFree(p_device.g_z));
	cudaVerify(hipFree(p_device.z0));
	cudaVerify(hipFree(p_device.dzdt));
	cudaVerify(hipFree(p_device.vz));
#if XSPH
	cudaVerify(hipFree(p_device.xsphvz));
#endif
	cudaVerify(hipFree(p_device.az));
	cudaVerify(hipFree(p_device.g_az));
#endif

#if TENSORIAL_CORRECTION
	cudaVerify(hipFree(p_device.tensorialCorrectionMatrix));
	//cudaVerify(hipFree(p_device.tensorialCorrectiondWdrr));
#endif

#if SHEPARD_CORRECTION
	cudaVerify(hipFree(p_device.shepard_correction));
#endif

#if INTEGRATE_ENERGY
	cudaVerify(hipHostFree(p_host.dedt));
	cudaVerify(hipFree(p_device.dedt));
#endif

	cudaVerify(hipHostFree(p_host.drhodt));
	cudaVerify(hipFree(p_device.drhodt));

#if INTEGRATE_SML
	cudaVerify(hipFree(p_device.dhdt));
#endif
#if SML_CORRECTION
	cudaVerify(hipFree(p_device.sml_omega));
#endif

#if NAVIER_STOKES
	cudaVerify(hipFree(p_device.Tshear));
	cudaVerify(hipHostFree(p_host.Tshear));
	cudaVerify(hipFree(p_device.eta));
#endif
#if SOLID
	cudaVerify(hipFree(p_device.S));
    cudaVerify(hipHostFree(p_host.ep));
	cudaVerify(hipFree(p_device.dSdt));
	cudaVerify(hipHostFree(p_host.S));
	cudaVerify(hipHostFree(p_host.dSdt));
	cudaVerify(hipFree(p_device.local_strain));
	cudaVerify(hipHostFree(p_host.local_strain));
    cudaVerify(hipFree(p_device.plastic_f));
	cudaVerify(hipFree(p_device.sigma));
    cudaVerify(hipFree(p_device.ep));
    cudaVerify(hipFree(p_device.edotp));
#endif
#if ARTIFICIAL_STRESS
	cudaVerify(hipFree(p_device.R));
#endif

#if JC_PLASTICITY
	cudaVerify(hipFree(p_device.T));
	cudaVerify(hipFree(p_device.dTdt));
	cudaVerify(hipFree(p_device.jc_f));
#endif

#if GHOST_BOUNDARIES
	cudaVerify(hipFree(p_device.real_partner));
#endif

#if FRAGMENTATION
	cudaVerify(hipHostFree(p_host.d));
	cudaVerify(hipFree(p_device.d));
	cudaVerify(hipFree(p_device.damage_total));
	cudaVerify(hipFree(p_device.dddt));
	cudaVerify(hipHostFree(p_host.dddt));
	cudaVerify(hipHostFree(p_host.numFlaws));
	cudaVerify(hipFree(p_device.numFlaws));
	cudaVerify(hipHostFree(p_host.numActiveFlaws));
	cudaVerify(hipFree(p_device.numActiveFlaws));
	cudaVerify(hipHostFree(p_host.flaws));
	cudaVerify(hipFree(p_device.flaws));
# if PALPHA_POROSITY
	cudaVerify(hipFree(p_device.damage_porjutzi));
	cudaVerify(hipFree(p_device.cs_old));
	cudaVerify(hipFree(p_device.ddamage_porjutzidt));
# endif
#endif


#if PALPHA_POROSITY
	cudaVerify(hipFree(p_device.alpha_jutzi));
	cudaVerify(hipFree(p_device.alpha_jutzi_old));
	cudaVerify(hipFree(p_device.pold));
	cudaVerify(hipFree(p_device.dalphadt));
	cudaVerify(hipFree(p_device.dalphadp));
	cudaVerify(hipFree(p_device.dp));
	cudaVerify(hipFree(p_device.dalphadrho));
	cudaVerify(hipFree(p_device.f));
	cudaVerify(hipFree(p_device.delpdelrho));
	cudaVerify(hipFree(p_device.delpdele));
#endif

#if SIRONO_POROSITY
    cudaVerify(hipFree(p_device.compressive_strength));
    cudaVerify(hipFree(p_device.tensile_strength));
    cudaVerify(hipFree(p_device.shear_strength));
    cudaVerify(hipFree(p_device.K));
    cudaVerify(hipFree(p_device.rho_0prime));
    cudaVerify(hipFree(p_device.rho_c_plus));
    cudaVerify(hipFree(p_device.rho_c_minus));
    cudaVerify(hipFree(p_device.flag_rho_0prime));
    cudaVerify(hipFree(p_device.flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipFree(p_device.alpha_epspor));
    cudaVerify(hipFree(p_device.dalpha_epspordt));
    cudaVerify(hipFree(p_device.epsilon_v));
    cudaVerify(hipFree(p_device.depsilon_vdt));
#endif

	cudaVerify(hipHostFree(p_host.x));
	cudaVerify(hipHostFree(p_host.vx));
	cudaVerify(hipHostFree(p_host.ax));
    cudaVerify(hipHostFree(p_host.g_ax));
#if DIM > 1
	cudaVerify(hipHostFree(p_host.y));
	cudaVerify(hipHostFree(p_host.vy));
	cudaVerify(hipHostFree(p_host.ay));
    cudaVerify(hipHostFree(p_host.g_ay));
#endif
	cudaVerify(hipHostFree(p_host.m));
	cudaVerify(hipHostFree(p_host.h));
	cudaVerify(hipHostFree(p_host.rho));
	cudaVerify(hipHostFree(p_host.p));
	cudaVerify(hipHostFree(p_host.e));
	cudaVerify(hipHostFree(p_host.cs));
	cudaVerify(hipHostFree(p_host.noi));
	cudaVerify(hipHostFree(interactions_host));
	cudaVerify(hipHostFree(p_host.depth));
	cudaVerify(hipHostFree(p_host.materialId));
	cudaVerify(hipHostFree(childList_host));
#if MORE_OUTPUT
	cudaVerify(hipHostFree(p_host.p_min));
	cudaVerify(hipHostFree(p_host.p_max));
	cudaVerify(hipHostFree(p_host.rho_min));
	cudaVerify(hipHostFree(p_host.rho_max));
	cudaVerify(hipHostFree(p_host.e_min));
	cudaVerify(hipHostFree(p_host.e_max));
	cudaVerify(hipHostFree(p_host.cs_min));
	cudaVerify(hipHostFree(p_host.cs_max));
#endif
#if INVISCID_SPH
	cudaVerify(hipHostFree(p_host.beta));
	cudaVerify(hipHostFree(p_host.beta_old));
	cudaVerify(hipHostFree(p_host.divv_old));
#endif
#if PALPHA_POROSITY
	cudaVerify(hipHostFree(p_host.alpha_jutzi));
	cudaVerify(hipHostFree(p_host.alpha_jutzi_old));
	cudaVerify(hipHostFree(p_host.dalphadt));
	cudaVerify(hipHostFree(p_host.pold));
# if FRAGMENTATION
    cudaVerify(hipHostFree(p_host.damage_porjutzi));
    cudaVerify(hipHostFree(p_host.ddamage_porjutzidt));
# endif
#endif

#if SIRONO_POROSITY
    cudaVerify(hipHostFree(p_host.compressive_strength));
    cudaVerify(hipHostFree(p_host.tensile_strength));
    cudaVerify(hipHostFree(p_host.shear_strength));
    cudaVerify(hipHostFree(p_host.rho_0prime));
    cudaVerify(hipHostFree(p_host.rho_c_plus));
    cudaVerify(hipHostFree(p_host.rho_c_minus));
    cudaVerify(hipHostFree(p_host.K));
    cudaVerify(hipHostFree(p_host.flag_rho_0prime));
    cudaVerify(hipHostFree(p_host.flag_plastic));
#endif

#if EPSALPHA_POROSITY
    cudaVerify(hipHostFree(p_host.alpha_epspor));
    cudaVerify(hipHostFree(p_host.epsilon_v));
#endif

#if JC_PLASTICITY
	cudaVerify(hipHostFree(p_host.T));
#endif
#if DIM > 2
	cudaVerify(hipHostFree(p_host.z));
	cudaVerify(hipHostFree(p_host.vz));
	cudaVerify(hipHostFree(p_host.az));
    cudaVerify(hipHostFree(p_host.g_az));
#endif

    free_aneos_memory();

    return rc;
}
